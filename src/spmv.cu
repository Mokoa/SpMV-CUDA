#include "hip/hip_runtime.h"
#include "spmv.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void spmvSimple(float* y, const float *A, const int *IA, const int *JA, const float *x)
{
        int row = threadIdx.x + blockDim.x * blockIdx.x;
        y[row] = 0;
        int j;
        for (j = IA[row]; j < IA[row+1]; ++j)
                y[row] += A[j]*x[JA[j]];

        __syncthreads();
}

void cpuSpMV(float *y, float *A, int *IA, int *JA, const int M, const float *x)
{
        int i, j;
        float sum;
        for (i = 0; i < M; ++i)
        {
                sum = 0;
                for (j = IA[i]; j < IA[i+1]; ++j)
                {
                        sum += A[j]*x[JA[j]];
                }
                y[i] = sum;
        }
}

void printArray(const float* arr, const int l)
{
        int i;
        printf("[");
        for (i = 0; i < l; ++i)
        {
                printf("%g", arr[i]);
                if (i != l-1)
                        printf("; ");
        }
        printf("];\n");
}

void printArray(const int* arr, const int l)
{
        int i;
        printf("[ ");
        for (i = 0; i < l; ++i)
        {
                printf("%i", arr[i]);
                if (i != l-1)
                        printf("; ");
        }
        printf("];\n");
}

bool areEqualRMSE(const float *a, const float *b, const int N)
{
        double RMSE_THRESHOLD = 1e-3;
        double sq_err_sum = 0; double rmse;
        int i;
        for (i = 0; i < N; ++i)
        {
                sq_err_sum += (a[i] - b[i])*(a[i] - b[i]);
        }
        rmse = sqrt(sq_err_sum/N);
        
        if (rmse > RMSE_THRESHOLD)
                printf("RMSE = %g\n", rmse);

        return rmse < RMSE_THRESHOLD;
}

void printSpMatrix(const float* A, const int* IA, const int* JA, const int M, const int N)
{
        int v = 0;

        int i, j;
        for (i = 0; i < M; ++i)
        {
                for (j = 0; j < N; ++j)
                {
                        if(v < IA[i+1] && j == JA[v])
                        {
                                printf("%g\t", A[v]);
                                v++;
                        }
                        else
                        {
                                printf("0\t");
                        }
                }
                printf("\n");
        }
}

void generateSquareSpMatrix(float **A_p, int **IA_p, int **JA_p, int *NNZ_p, const int N, const double p_diag, const double p_nondiag)
{
        // estimate size of A, JA arrays because they vary between realization
        // but are same for a given realization
        int estSize = N*p_diag + N*(N-1)*p_nondiag;
        
        //printf("Estimate size of A: %i\n", estSize);
        //printf("Size of IA: %i\n", N+1);
        //printf("Estimate size of JA: %i\n", estSize);
        
        // allocate IA because size is fixed (size of IA = N + 1)
        *IA_p = (int *)malloc(sizeof(int)*(N+1));
        
        // define buffer space for undetermined arrays
        int bufferSize = (int)ceil(1.33*estSize);
        //printf("Buffer size = %i\n", bufferSize);
        
        // allocate buffer*estSize for A & JA so we can probably fit everything in those
        float* A_temp = (float *)malloc(sizeof(float)*bufferSize);
        int* JA_temp = (int *)malloc(sizeof(float)*bufferSize);
        
        double randProb; float randNum;

        // Setup inital conditions for sparse matrix
        *NNZ_p = 0; (*IA_p)[0] = 0;

        int i,j;
        for (i = 0; i < N; ++i)
        {
                (*IA_p)[i+1] = (*IA_p)[i];
                
                for (j = 0; j < N; ++j)
                {
                        randProb = ((double)rand())/RAND_MAX;
                        if (i == j) // on diagonal - use p_diag
                        {
                                if (randProb < p_diag) // insert non-zero element
                                {
                                        if((*NNZ_p) == bufferSize) // Placing element will exceed allowed buffer!
                                        {
                                                resizeSpMatrixArraysAndCopy(&A_temp, &JA_temp, &bufferSize, 1.33); // resize arrays so we can insert element!
                                                //printf("Error: Exceeded allowed buffer size. Failed to create sparse matrix!\n");
                                                //return;
                                        }
                                        
                                        // Place random non-zero element into sparse matrix
                                        randNum = getRandomFloat(0, 1);
                                        A_temp[(*NNZ_p)] = randNum;
                                        JA_temp[(*NNZ_p)] = j;
                                        (*IA_p)[i+1]++;
                                        (*NNZ_p)++;
                                }
                        }
                        else // not on diagonal - use p_nondiag
                        {
                                if (randProb < p_nondiag)
                                {
                                        if((*NNZ_p) == bufferSize) // Placing element will exceed allowed buffer!
                                        {
                                                resizeSpMatrixArraysAndCopy(&A_temp, &JA_temp, &bufferSize, 1.33); // resize arrays so we can insert element!
                                                //printf("Error: Exceeded allowed buffer size. Failed to create sparse matrix!\n");
                                                //return;
                                        }
                                        
                                        // Place random non-zero element into sparse matrix
                                        randNum = getRandomFloat(0, 1);;
                                        A_temp[(*NNZ_p)] = randNum;
                                        JA_temp[(*NNZ_p)] = j;
                                        (*IA_p)[i+1]++;
                                        (*NNZ_p)++;
                                        
                                }
                        }
                }
        }

        //printf("A_temp: "); printArray(A_temp, bufferSize);
        //printf("IA: "); printArray(*IA_p, N+1);
        //printf("JA_temp: "); printArray(JA_temp, bufferSize);

        // By this point we have not exceeded memory limit so lets create
        // actual A and IA array now that we have determined the size
        *A_p = (float *)malloc(sizeof(float)*(*NNZ_p));
        *JA_p = (int *)malloc(sizeof(float)*(*NNZ_p));
        
        // Add elements from temp arrays to actual arrays
        for (i = 0; i < (*NNZ_p); ++i)
        {
                (*A_p)[i] = A_temp[i];
                (*JA_p)[i] = JA_temp[i];
        }
        
        //printf("A: "); printArray(*A_p, *NNZ_p);
        //printf("IA: "); printArray(*IA_p, N+1);
        //printf("JA: "); printArray(*JA_p, *NNZ_p);
        //printf("NNZ: %i\n", *NNZ_p);
       
        // free no longer used temp arrays
        free(A_temp); A_temp = NULL;
        free(JA_temp); JA_temp = NULL;
        
        return;
}

void resizeSpMatrixArraysAndCopy(float **A_temp_p, int **JA_temp_p, int *bufferSize_p, double RESIZE_FACTOR)
{

        printf("Executing resize!!\n");
        if (RESIZE_FACTOR <= 1) // RESIZE_FACTOR should not be less than one!
                RESIZE_FACTOR = 1.33; // if so, set to default value of 1.33

        int oldLength = (*bufferSize_p);
        int newLength = (int)ceil((*bufferSize_p)*RESIZE_FACTOR);
        float *A_temp_new;
        int *JA_temp_new;

        // allocate the new resized memory
        A_temp_new = (float *)malloc(sizeof(float)*newLength);
        JA_temp_new = (int *)malloc(sizeof(int)*newLength);

        // copy old elements into new array
        int i;
        for (i = 0; i < oldLength; ++i)
        {
                A_temp_new[i] = (*A_temp_p)[i];
                JA_temp_new[i] = (*JA_temp_p)[i];
        }

        // free memory from old arrays
        free(*A_temp_p);
        free(*JA_temp_p);

        // update pointers
        *A_temp_p = A_temp_new; A_temp_new = NULL;
        *JA_temp_p = JA_temp_new; A_temp_new = NULL;

        // update bufferSize
        *bufferSize_p = newLength;
}

float getRandomFloat(const float min, const float max)
{
        return ((((float)rand())/RAND_MAX)*(max-min)+min);
}

void fillDenseVector(float* v, const int N)
{
        int i;
        for (i = 0; i < N; ++i)
                v[i] = getRandomFloat(0, 1);
}










