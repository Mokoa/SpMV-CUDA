#include "spmv.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int main()
{
        // PARAMETERS
        double p_diag = 0.9;
        double p_nondiag = 0.1;
        float *A_cpu, *A_gpu, *x_cpu, *x_gpu, *y_cpu, *y_gpu, *y_correct;
        int *IA_cpu, *IA_gpu, *JA_cpu, *JA_gpu;
        int NNZ;

        int expNmin = 1;
        int expNmax = 15;
        int Nmin = (1 << expNmin);
        int Nmax = (1 << expNmax);
        int L = expNmax-expNmin+1; //printf("L = %i\n", L);
        float *t_arr = (float *)malloc(sizeof(float)*L);
        int *N_arr = (int *)malloc(sizeof(float)*L);
        int i; int idx = 0;
        for (i = 0; i < L; ++i)
                N_arr[i] = (1 << (i+1));

        // seed random number generator
        time_t t; srand((unsigned) time(&t));

        const int NUM_ITERS = 1;

        // Define cuda events
        float milliseconds;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        int N, iter; double elapsed;
        for (N = Nmin; N <= Nmax; N=N*2)
        {
                elapsed = 0;
                for (iter = 0; iter < NUM_ITERS; ++iter)
                {
                        // Create sparse matrix
                        generateSquareSpMatrix(&A_cpu, &IA_cpu, &JA_cpu, &NNZ, N, p_diag, p_nondiag); // allocates!

                        // Generate dense vector x
                        x_cpu = (float *)malloc(sizeof(float)*N);
                        fillDenseVector(x_cpu, N);
                        
                        // Define output vector y and y_correct
                        y_cpu = (float *)malloc(sizeof(float)*N);
                        y_correct = (float *)malloc(sizeof(float)*N);

                        // Setup memory on the GPU
                        hipMalloc((void**) &A_gpu, NNZ*sizeof(float));
                        hipMalloc((void**) &IA_gpu, (N+1)*sizeof(int)); // N = M
                        hipMalloc((void**) &JA_gpu, NNZ*sizeof(int));
                        hipMalloc((void**) &x_gpu, N*sizeof(float));
                        hipMalloc((void**) &y_gpu, N*sizeof(float)); // N = M
        
                        // Transfer to device
                        hipMemcpy(A_gpu, A_cpu, NNZ*sizeof(float), hipMemcpyHostToDevice);
                        hipMemcpy(IA_gpu, IA_cpu, (N+1)*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(JA_gpu, JA_cpu, NNZ*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(x_gpu, x_cpu, N*sizeof(float), hipMemcpyHostToDevice);
                        
                        // CUDA kernel parameters
                        int threadsPerBlock, blocksPerGrid;
                        if (N < 1024)
                        {
                                threadsPerBlock = N;
                                blocksPerGrid = 1;
                        }
                        else
                        {
                                threadsPerBlock = 1024;
                                blocksPerGrid = N / 1024;
                        }

                        // Start cudaEvent timing
                        hipEventRecord(start);
                        
                        // CUDA Simple SpMV Kernel
                        spmvSimple<<<blocksPerGrid, threadsPerBlock>>>(y_gpu, A_gpu, IA_gpu, JA_gpu, x_gpu);
                       
                        // Stop cudaEvent timing
                        hipEventRecord(stop);
                        hipEventSynchronize(stop);

                        // Check to make sure that cuda kernel was successful
                        hipError_t err = hipGetLastError();
                        if (err != hipSuccess)
                                printf("Error: %s\n", hipGetErrorString(err));

                        // Print result
                        milliseconds = 0;
                        hipEventElapsedTime(&milliseconds, start, stop);
                        elapsed += milliseconds;

                        // Transfer result back to host
                        hipMemcpy(y_cpu, y_gpu, N*sizeof(float), hipMemcpyDeviceToHost);

                        // Test correctness of CUDA kernel vs "golden" cpu spmv function
                        cpuSpMV(y_correct, A_cpu, IA_cpu, JA_cpu, N, x_cpu);
                        if (!areEqualRMSE(y_correct, y_cpu, N))
                                printf("Not correct result for a (%ix%i)*(%ix1) spmv multiplication\n", N, N, N);

                        // Free memory
                        free(A_cpu);
                        free(IA_cpu);
                        free(JA_cpu);
                        free(x_cpu);
                        free(y_cpu);
                        free(y_correct);
                        hipFree(A_gpu);
                        hipFree(IA_gpu);
                        hipFree(JA_gpu);
                        hipFree(x_gpu);
                        hipFree(y_gpu);
                }
                //printf("Average performace of N = %i SpMV over %i iterations: %g ms\n", N, NUM_ITERS, elapsed/NUM_ITERS);
                t_arr[idx++] = (float)elapsed/NUM_ITERS;
        }

        printf("N = "); printArray(N_arr, L);
        printf("t = "); printArray(t_arr, L);
        
        free(t_arr);
        free(N_arr);

        hipDeviceReset();
	return 0;
}
