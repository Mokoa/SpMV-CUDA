#include "spmv.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

int main()
{
        printf("\n============================== TEST: SPMV TIMING ==========================================\n\n");

        printf("Running spmvChocolate with BLOCK_SIZE = %i\n", BLOCK_SIZE);

        // PARAMETERS
        double p_diag = 0.9;
        double p_nondiag = 0.1;
        float *A_cpu, *A_gpu, *x_cpu, *x_gpu, *y_cpu, *y_gpu, *y_correct;
        int *IA_cpu, *IA_gpu, *JA_cpu, *JA_gpu;
        int NNZ;

        int expMmin = 10;
        int expMmax = 15;
        int Mmin = (1 << expMmin);
        int Mmax = (1 << expMmax);
        int L = expMmax-expMmin+1; //printf("L = %i\n", L);
        float *t_arr = (float *)malloc(sizeof(float)*L);
        int *M_arr = (int *)malloc(sizeof(float)*L);
        int i; int idx = 0;
        for (i = 0; i < L; ++i)
                M_arr[i] = (1 << (i+expMmin));

        // seed random number generator
        time_t t; srand((unsigned) time(&t));

        const int NUM_ITERS = 1;

        // Define cuda events for GPU timing
        float milliseconds;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Setup CPU timing for cpuSpMV
        //struct timeval t1, t2;
        
        int M, N, iter; double elapsed;
        for (M = Mmin; M <= Mmax; M=M*2)
        {
                elapsed = 0;
                for (iter = 0; iter < NUM_ITERS; ++iter)
                {
                        // Create sparse matrix
                        generateSquareSpMatrix(&A_cpu, &IA_cpu, &JA_cpu, &NNZ, M, p_diag, p_nondiag); // allocates!
                        N = M; // for square matrices
                        // Generate dense vector x
                        x_cpu = (float *)malloc(sizeof(float)*N);
                        fillDenseVector(x_cpu, N);
                        
                        // Define output vector y and y_correct
                        y_cpu = (float *)malloc(sizeof(float)*M);
                        y_correct = (float *)malloc(sizeof(float)*M);

                        // Setup memory on the GPU
                        hipMalloc((void**) &A_gpu, NNZ*sizeof(float));
                        hipMalloc((void**) &IA_gpu, (M+1)*sizeof(int));
                        hipMalloc((void**) &JA_gpu, NNZ*sizeof(int));
                        hipMalloc((void**) &x_gpu, N*sizeof(float));
                        hipMalloc((void**) &y_gpu, M*sizeof(float));
        
                        // Transfer to device
                        hipMemcpy(A_gpu, A_cpu, NNZ*sizeof(float), hipMemcpyHostToDevice);
                        hipMemcpy(IA_gpu, IA_cpu, (M+1)*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(JA_gpu, JA_cpu, NNZ*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(x_gpu, x_cpu, N*sizeof(float), hipMemcpyHostToDevice);
                        
                        // CUDA kernel parameters
                        int dB, dG;
                        dB = BLOCK_SIZE;
                        dG = M / BLOCK_SIZE;
                        
                        // Do CPU timing
                        //gettimeofday(&t1, NULL);
                        //spmvCPU(y_cpu, A_cpu, IA_cpu, JA_cpu, M, x_cpu);
                        //gettimeofday(&t2, NULL);
                        //elapsed += (t2.tv_sec-t1.tv_sec)*1000.0 + (t2.tv_usec-t1.tv_usec)/1000.0; // in ms

                        // Start cudaEvent timing
                        hipEventRecord(start);
                        
                        // CUDA Vanilla SpMV Kernel
                        spmvVanilla<<< dG, dB >>>(y_gpu, A_gpu, IA_gpu, JA_gpu, M,  x_gpu);

                        // CUDA Chocolate SpMV Kernel
                        //spmvChocolate<<< dG, dB >>>(y_gpu, A_gpu, IA_gpu, JA_gpu, M, x_gpu);
                       
                        // Stop cudaEvent timing
                        hipEventRecord(stop);
                        hipEventSynchronize(stop);

                        // Check to make sure that cuda kernel was successful
                        hipError_t err = hipGetLastError();
                        if (err != hipSuccess)
                                printf("Error: %s\n", hipGetErrorString(err));

                        // Record timing result
                        milliseconds = 0;
                        hipEventElapsedTime(&milliseconds, start, stop);
                        elapsed += milliseconds;

                        // Transfer result back to host
                        hipMemcpy(y_cpu, y_gpu, M*sizeof(float), hipMemcpyDeviceToHost);

                        // Test correctness of CUDA kernel vs "golden" cpu spmv function
                        spmvCPU(y_correct, A_cpu, IA_cpu, JA_cpu, M, x_cpu);
                        if (!areEqualRMSE(y_correct, y_cpu, M))
                                printf("Not correct result for a (%ix%i)*(%ix1) spmv multiplication\n", M, N, N);

                        // Free memory
                        free(A_cpu);
                        free(IA_cpu);
                        free(JA_cpu);
                        free(x_cpu);
                        free(y_cpu);
                        free(y_correct);
                        hipFree(A_gpu);
                        hipFree(IA_gpu);
                        hipFree(JA_gpu);
                        hipFree(x_gpu);
                        hipFree(y_gpu);
                }
                t_arr[idx++] = (float)elapsed/NUM_ITERS;
        }

        printf("Results averaged over %i iterations with time in ms:\n", NUM_ITERS);
        printf("M = "); printArray(M_arr, L);
        printf("t = "); printArray(t_arr, L);
        
        free(t_arr);
        free(M_arr);

        hipDeviceReset();
	
        
        printf("\n===========================================================================================\n\n");

        return 0;
}
