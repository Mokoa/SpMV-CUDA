#include "spmv.h"
#include <stdio.h>
#include <stdlib.h>

int main()
{

        printf("\n============================== TEST: spmvStrawberry =======================================\n\n");

        // PARAMETERS
        double p_diag = 0.9;
        double p_nondiag = 0.1;
        float *A_cpu, *A_gpu, *x_cpu, *x_gpu;
        float *y_cpu_correct, *y_cpu_strawberry;
        float *y_gpu_strawberry;
        int *IA_cpu, *IA_gpu, *JA_cpu, *JA_gpu;
        int NNZ;

        // seed random number generator
        time_t t; srand((unsigned) time(&t));

        // Define cuda events
        int M, N;
        for (M = (1 << 5); M <= (1 << 15); M=M*2)
        {
                // Create sparse matrix
                generateSquareSpMatrix(&A_cpu, &IA_cpu, &JA_cpu, &NNZ, M, p_diag, p_nondiag); // allocates!
                N = M; // for square matrices

                // Generate dense vector x
                x_cpu = (float *)malloc(sizeof(float)*N);
                fillDenseVector(x_cpu, N);
                
                // Define output vector y
                y_cpu_correct = (float *)malloc(sizeof(float)*M);
                y_cpu_strawberry = (float *)malloc(sizeof(float)*M);

                // Setup memory on the GPU
                hipMalloc((void**) &A_gpu, NNZ*sizeof(float));
                hipMalloc((void**) &IA_gpu, (M+1)*sizeof(int)); // N = M
                hipMalloc((void**) &JA_gpu, NNZ*sizeof(int));
                hipMalloc((void**) &x_gpu, N*sizeof(float));
                hipMalloc((void**) &y_gpu_strawberry, M*sizeof(float)); // N = M
        
                // Transfer to device
                hipMemcpy(A_gpu, A_cpu, NNZ*sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(IA_gpu, IA_cpu, (M+1)*sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(JA_gpu, JA_cpu, NNZ*sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(x_gpu, x_cpu, N*sizeof(float), hipMemcpyHostToDevice);
                
                // CUDA kernel parameters
                int dB, dG;
                if (M < 1024)
                {
                        dB = M;
                        dG = M / dB * 32;
                }
                else
                {
                        dB = BLOCK_SIZE;
                        dG = M / dB * 32;
                }

                // CPU SpMV kernel
                spmvCPU(y_cpu_correct, A_cpu, IA_cpu, JA_cpu, M, x_cpu);

                // Strawberry SpMV CUDA kernel
                spmvStrawberry<<< dG, dB, 32768 >>>(y_gpu_strawberry, A_gpu, IA_gpu, JA_gpu, M, x_gpu);

                // Transfer result back to host
                hipMemcpy(y_cpu_strawberry, y_gpu_strawberry, M*sizeof(float), hipMemcpyDeviceToHost);

                // Test correctness of SpMV CUDA kernel flavors  vs "golden" cpu spmv function
                if (areEqualRMSE(y_cpu_correct, y_cpu_strawberry, N))
                        printf("spmvStrawberry is correct for a (%ix%i)*(%ix1) SpMV multiplication\n", M, N, N);
                else
                        printf("spmvStrawberry is NOT correct for a (%ix%i)*(%ix1) SpMV multiplication\n", M, N, N);
                printRMSE(y_cpu_correct, y_cpu_strawberry, N);
                printf("\n");

                // Free memory
                free(A_cpu);
                free(IA_cpu);
                free(JA_cpu);
                free(x_cpu);
                free(y_cpu_correct);
                free(y_cpu_strawberry);
                hipFree(A_gpu);
                hipFree(IA_gpu);
                hipFree(JA_gpu);
                hipFree(x_gpu);
                hipFree(y_gpu_strawberry);
                
        }
        
        hipDeviceReset();
	
        
        printf("\n===========================================================================================\n\n");
        return 0;
}
