#include "spmv.h"
#include <stdio.h>
#include <stdlib.h>

int main()
{
        // PARAMETERS
        double p_diag = 0.9;
        double p_nondiag = 0.001;
        float *A_cpu, *A_gpu, *x_cpu, *x_gpu, *y_cpu, *y_gpu, *y_correct;
        int *IA_cpu, *IA_gpu, *JA_cpu, *JA_gpu;
        int NNZ;

        // seed random number generator
        time_t t; srand((unsigned) time(&t));

        const int NUM_ITERS = 1;

        // Define cuda events
        int N, iter;
        for (N = 2; N <= (1 << 15); N=N*2)
        {
                for (iter = 0; iter < NUM_ITERS; ++iter)
                {
                        // Create sparse matrix
                        generateSquareSpMatrix(&A_cpu, &IA_cpu, &JA_cpu, &NNZ, N, p_diag, p_nondiag); // allocates!

                        // Generate dense vector x
                        x_cpu = (float *)malloc(sizeof(float)*N);
                        fillDenseVector(x_cpu, N);
                        
                        // Define output vector y
                        y_cpu = (float *)malloc(sizeof(float)*N);
                        y_correct = (float *)malloc(sizeof(float)*N);

                        // Setup memory on the GPU
                        hipMalloc((void**) &A_gpu, NNZ*sizeof(float));
                        hipMalloc((void**) &IA_gpu, (N+1)*sizeof(int)); // N = M
                        hipMalloc((void**) &JA_gpu, NNZ*sizeof(int));
                        hipMalloc((void**) &x_gpu, N*sizeof(float));
                        hipMalloc((void**) &y_gpu, N*sizeof(float)); // N = M
        
                        // Transfer to device
                        hipMemcpy(A_gpu, A_cpu, NNZ*sizeof(float), hipMemcpyHostToDevice);
                        hipMemcpy(IA_gpu, IA_cpu, (N+1)*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(JA_gpu, JA_cpu, NNZ*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(x_gpu, x_cpu, N*sizeof(float), hipMemcpyHostToDevice);
                        
                        // CUDA kernel parameters
                        int threadsPerBlock, blocksPerGrid;
                        if (N < 1024)
                        {
                                threadsPerBlock = N;
                                blocksPerGrid = 1;
                        }
                        else
                        {
                                threadsPerBlock = 1024;
                                blocksPerGrid = N / 1024;
                        }

                        // Simple SpMV CUDA kernel
                        spmvSimple<<<blocksPerGrid, threadsPerBlock>>>(y_gpu, A_gpu, IA_gpu, JA_gpu, x_gpu);
                        
                        // Transfer result back to host
                        hipMemcpy(y_cpu, y_gpu, N*sizeof(float), hipMemcpyDeviceToHost);

                        // Test correctness of CUDA kernel vs "golden" cpu spmv function
                        cpuSpMV(y_correct, A_cpu, IA_cpu, JA_cpu, N, x_cpu);
                        if (areEqualRMSE(y_correct, y_cpu, N))
                                printf("GPU kernel result is correct for a (%ix%i)*(%ix1) spmv multiplication\n", N, N, N);
                        else
                                printf("GPU kernel result is NOT correct for a (%ix%i)*(%ix1) spmv multiplication\n", N, N, N);


                        // Free memory
                        free(A_cpu);
                        free(IA_cpu);
                        free(JA_cpu);
                        free(x_cpu);
                        free(y_cpu);
                        free(y_correct);
                        hipFree(A_gpu);
                        hipFree(IA_gpu);
                        hipFree(JA_gpu);
                        hipFree(x_gpu);
                        hipFree(y_gpu);
                }
        }
        
        hipDeviceReset();
	return 0;
}
