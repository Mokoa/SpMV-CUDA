#include "spmv.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

int main()
{
        // Set CUDA to prefer L1 cache over shared memory
        // Will set L1 cache to 48K and shared memory to 16K if possible
        hipDeviceSetCacheConfig(hipFuncCachePreferL1);

        // Do gettimeofday timing for estimating execution time of main
        struct timeval t1_main, t2_main;//, t1, t2;
        double elapsedMain;
        gettimeofday(&t1_main, NULL);

        printf("Running main using spmvStrawberry with block size %i\n", BLOCK_SIZE);

        // PARAMETERS
        double p_diag = 0.9;
        double p_nondiag = 0.1;
        float *A_cpu, *A_gpu, *x_cpu, *x_gpu, *y_cpu, *y_gpu, *y_correct;
        int *IA_cpu, *IA_gpu, *JA_cpu, *JA_gpu;
        int NNZ;

        int expMmin = 10;
        int expMmax = 15;
        int Mmin = (1 << expMmin);
        int Mmax = (1 << expMmax);
        int L = expMmax-expMmin+1; //printf("L = %i\n", L);
        float *t_arr = (float *)malloc(sizeof(float)*L);
        int *M_arr = (int *)malloc(sizeof(float)*L);
        int i; int idx = 0;
        for (i = 0; i < L; ++i)
                M_arr[i] = (1 << (i+expMmin));

        // seed random number generator
        time_t t; srand((unsigned) time(&t));

        const int NUM_ITERS = 1;

        // Define cuda events for GPU timing
        float milliseconds;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        // Define CudaError
        hipError_t err;

        // Define CUDA kernel parameters
        //int dB, dG;
        int dB_strawberry, dG_strawberry;

        int M, N, iter; double elapsed;
        for (M = Mmin; M <= Mmax; M=M*2)
        {
                elapsed = 0;
                for (iter = 0; iter < NUM_ITERS; ++iter)
                {
                        // Create sparse matrix
                        generateSquareSpMatrix(&A_cpu, &IA_cpu, &JA_cpu, &NNZ, M, p_diag, p_nondiag); // allocates!
                        N = M; // due to generation of square sparse matrix
                        
                        // Generate dense vector x
                        x_cpu = (float *)malloc(sizeof(float)*N);
                        fillDenseVector(x_cpu, N);
                        
                        // Define output vector y and y_correct
                        y_cpu = (float *)malloc(sizeof(float)*M);
                        y_correct = (float *)malloc(sizeof(float)*M);

                        // Setup memory on the GPU
                        hipMalloc((void**) &A_gpu, NNZ*sizeof(float));
                        hipMalloc((void**) &IA_gpu, (M+1)*sizeof(int)); // N = M
                        hipMalloc((void**) &JA_gpu, NNZ*sizeof(int));
                        hipMalloc((void**) &x_gpu, N*sizeof(float));
                        hipMalloc((void**) &y_gpu, M*sizeof(float)); // N = M
        
                        // Transfer to device
                        hipMemcpy(A_gpu, A_cpu, NNZ*sizeof(float), hipMemcpyHostToDevice);
                        hipMemcpy(IA_gpu, IA_cpu, (M+1)*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(JA_gpu, JA_cpu, NNZ*sizeof(int), hipMemcpyHostToDevice);
                        hipMemcpy(x_gpu, x_cpu, N*sizeof(float), hipMemcpyHostToDevice);
                        
                        // Set CUDA kernel parameters
                        //dB = BLOCK_SIZE;
                        //dG = N / 1024;
                        dB_strawberry = BLOCK_SIZE;
                        dG_strawberry = M / BLOCK_SIZE * 32;
                        
                        // Do CPU timing
                        //gettimeofday(&t1, NULL);
                        //spmvCPU(y_cpu, A_cpu, IA_cpu, JA_cpu, M, x_cpu);
                        //gettimeofday(&t2, NULL);
                        //elapsed += (t2.tv_sec-t1.tv_sec)*1000.0 + (t2.tv_usec-t1.tv_usec)/1000.0; // in ms

                        // Start cudaEvent timing
                        hipEventRecord(start);
                        
                        // CUDA Vanilla SpMV Kernel
                        //spmvVanilla<<<dG, dB>>>(y_gpu, A_gpu, IA_gpu, JA_gpu, M, x_gpu);

                        // CUDA Chocolate SpMV Kernel
                        //spmvChocolate<<<dG, dB>>>(y_gpu, A_gpu, IA_gpu, JA_gpu, M, x_gpu);
                       
                        // CUDA Strawberry SpMV Kernel
                        spmvStrawberry<<< dG_strawberry, dB_strawberry >>>(y_gpu, A_gpu, IA_gpu, JA_gpu, M, x_gpu);

                        // Stop cudaEvent timing
                        hipEventRecord(stop);
                        hipEventSynchronize(stop);

                        // Check to make sure that cuda kernel was successful
                        err = hipGetLastError();
                        if (err != hipSuccess)
                                printf("Error: %s\n", hipGetErrorString(err));

                        // Record timing result
                        milliseconds = 0;
                        hipEventElapsedTime(&milliseconds, start, stop);
                        elapsed += milliseconds;

                        // Transfer result back to host
                        hipMemcpy(y_cpu, y_gpu, N*sizeof(float), hipMemcpyDeviceToHost);

                        // Test correctness of CUDA kernel vs "golden" cpu spmv function
                        spmvCPU(y_correct, A_cpu, IA_cpu, JA_cpu, N, x_cpu);
                        if (!areEqualRMSE(y_correct, y_cpu, N))
                        {
                                printf("Not correct result for a (%ix%i)*(%ix1) spmv multiplication\n", M, N, N);
                                printRMSE(y_correct, y_cpu, N);
                        }

                        // Free memory
                        free(A_cpu);
                        free(IA_cpu);
                        free(JA_cpu);
                        free(x_cpu);
                        free(y_cpu);
                        free(y_correct);
                        hipFree(A_gpu);
                        hipFree(IA_gpu);
                        hipFree(JA_gpu);
                        hipFree(x_gpu);
                        hipFree(y_gpu);
                }
                t_arr[idx++] = (float)elapsed/NUM_ITERS;
        }

        printf("Results averaged over %i iterations with time in ms:\n", NUM_ITERS);
        printf("M = "); printArray(M_arr, L);
        printf("t = "); printArray(t_arr, L);
        
        free(t_arr);
        free(M_arr);

        hipDeviceReset();

        gettimeofday(&t2_main, NULL);
        //elapsed += (t2.tv_sec-t1.tv_sec)*1000.0 + (t2.tv_usec-t1.tv_usec)/1000.0; // in ms
        elapsedMain = (t2_main.tv_sec-t1_main.tv_sec) + (t2_main.tv_usec-t1_main.tv_usec)/1000000.0;
        printf("Total time to execute main() : %g seconds\n", elapsedMain);
	return 0;
}
