#include "spmv.h"
#include <stdio.h>
#include <stdlib.h>

int main()
{
        // PARAMETERS
        double p_diag = 0.8;
        double p_nondiag = 0.05;
        int N = 4;
        float *A_cpu, *A_gpu, *x_cpu, *x_gpu, *y_cpu, *y_gpu;
        int *IA_cpu, *IA_gpu, *JA_cpu, *JA_gpu;
        int NNZ;

        // Define cuda events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Create sparse matrix
        SpMatrix S = generateSquareSpMatrix(N, p_diag, p_nondiag); // allocates!
        IA_cpu = S.IA; A_cpu = S.A; JA_cpu = S.JA; NNZ = S.NNZ;

        // Verify correctness by hand
        printf("Sparse Matrix S: \n"); printSpMatrix(S);
        printf("A: "); printArray(A_cpu, NNZ);
        printf("IA: "); printArray(IA_cpu, N+1);
        printf("JA: "); printArray(JA_cpu, NNZ);

        // Generate dense vector x
        x_cpu = (float *)malloc(sizeof(float)*N);
        fillDenseVector(x_cpu, N);
        printf("Dense vector x: "); printArray(x_cpu, N);
        
        // Define output vector y
        y_cpu = (float *)malloc(sizeof(float)*N);

        // Setup memory on the GPU
        hipMalloc((void**) &A_gpu, NNZ*sizeof(float));
        hipMalloc((void**) &IA_gpu, (N+1)*sizeof(int)); // N = M
        hipMalloc((void**) &JA_gpu, NNZ*sizeof(int));
        hipMalloc((void**) &x_gpu, N*sizeof(float));
        hipMalloc((void**) &y_gpu, N*sizeof(float)); // N = M
        
        
        // Transfer to device
        hipMemcpy(A_gpu, A_cpu, NNZ*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(IA_gpu, IA_cpu, (N+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(JA_gpu, JA_cpu, NNZ*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(x_gpu, x_cpu, N*sizeof(float), hipMemcpyHostToDevice);

        // Compute spmv multiplication
        hipEventRecord(start);
        //cpuSpMV(y, A, x);
        spmvSimple<<<1, N>>>(y_gpu, A_gpu, IA_gpu, JA_gpu, x_gpu); // supports only up to 1024
        hipEventRecord(stop);

        // Print result
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Elapsed time (ms) = %f\n", milliseconds);

        // Transfer to host
        hipMemcpy(y_cpu, y_gpu, N*sizeof(float), hipMemcpyDeviceToHost);
        printf("Output vector y: "); printArray(y_cpu, N);

        // Free memory
        free(A_cpu);
        free(IA_cpu);
        free(JA_cpu);
        free(x_cpu);
        free(y_cpu);
        hipFree(A_gpu);
        hipFree(IA_gpu);
        hipFree(JA_gpu);
        hipFree(x_gpu);
        hipFree(y_gpu);

        // Set dangling pointers to NULL
        S.A = NULL;
        S.IA = NULL;
        S.JA = NULL;

        hipDeviceReset();
	return 0;
}
